#include "hip/hip_runtime.h"
#include "fallback_allocator.cuh"

//#include <boost/chrono.hpp>
//#include <boost/log/common.hpp>
//#include <boost/log/expressions.hpp>
//#include <boost/log/utility/setup/file.hpp>
//#include <boost/log/utility/setup/console.hpp>
//#include <boost/log/utility/setup/common_attributes.hpp>
//#include <boost/log/attributes/timer.hpp>
//#include <boost/log/attributes/named_scope.hpp>
//#include <boost/log/sources/logger.hpp>
//#include <boost/log/core.hpp>
//#include <boost/log/expressions.hpp>

char *fallback_allocator::allocate(std::ptrdiff_t n){
	//boost::log::sources::logger lg;

	char *result = 0;

	      size_t mfree;
	      size_t mtotal;

	      //BOOST_LOG(lg) << "trying allocate:";

	      if(hipMemGetInfo 	( &mfree, &mtotal) ==  hipSuccess){
	    	  //BOOST_LOG(lg) << "memoryfree: " << mfree << " of memorytotal " << mtotal;

	    	  if(mfree < n){
	    		  //BOOST_LOG(lg) << "no GPU memory...";
	    	  }
	      }

	      // attempt to allocate device memory
	      if(hipMalloc(&result, n) == hipSuccess)
	      {
	    	  //BOOST_LOG(lg) << "allocated " << n << " bytes of device memory";

	        if(hipMemGetInfo 	( &mfree, &mtotal) ==  hipSuccess){
	        	//BOOST_LOG(lg) << "AFTER hipMalloc memoryfree: " << mfree << " of memorytotal " << mtotal;
		    }
	      }
	      else
	      {
	        // reset the last CUDA error
	        hipGetLastError();

	        // attempt to allocate pinned host memory
	        void *h_ptr = 0;
	        if(hipHostMalloc(&h_ptr, n) == hipSuccess)
	        {
	          // attempt to map host pointer into device memory space
	          if(hipHostGetDevicePointer(&result, h_ptr, 0) == hipSuccess)
	          {
	        	  //BOOST_LOG(lg) << "allocated " << n << " bytes of pinned host memory (fallback successful)";
	          }
	          else
	          {
	            // reset the last CUDA error
	            hipGetLastError();

	            // attempt to deallocate buffer
	            //BOOST_LOG(lg) << "failed to map host memory into device address space (fallback failed)";
	            hipHostFree(h_ptr);

	            throw std::bad_alloc();
	          }
	        }
	        else
	        {
	          // reset the last CUDA error
	          hipGetLastError();

	          //BOOST_LOG(lg) << "failed to allocate " << n << " bytes of memory (fallback failed)";

	          throw std::bad_alloc();
	        }
	      }
	      return result;
}

void fallback_allocator::deallocate(char *ptr, size_t n)
{
	void *raw_ptr = thrust::raw_pointer_cast(ptr);

	// determine where memory resides
	hipPointerAttribute_t	attributes;

	if(hipPointerGetAttributes(&attributes, raw_ptr) == hipSuccess){
		// free the memory in the appropriate way
		if(attributes.memoryType == hipMemoryTypeHost){
			hipHostFree(raw_ptr);
		}
		else{
			hipFree(raw_ptr);
		}
	}
}


